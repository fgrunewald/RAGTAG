#include "hip/hip_runtime.h"
/*******************************************||********************************************
                           Genetic algorithm optimizer of RAGTAG                               *
                                      genA.cu                                            *
Runs iterations of genetic algoirthm to optimize molecular mechanics dihedral parameters * 
                                                                                         *
              @author James Maier, Kellon Belfon, Chuan Tian                             *
              @lab Carlos Simmerling lab, Stony Brook University                         *
              @version 3.0 2019 Feb                                                      *
********************************************||*******************************************/
/*****************************************************************************************
* 	                ---------------LOAD LIBRARIES-------------                       *  
*****************************************************************************************/
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/generate.h>
#include <thrust/device_ptr.h>
#include <list>
#include <map>
#include "load.h"
#include "parse.h"
using namespace std;

/******** Number of threads for a given block, 256 block threads (index 0 to 255) *******/
const int BLOCK_SIZE=256;

#define HANDLE_ERROR(x) x;


/*****************************************************************************************
*                  Defining the six pivotal functions for the genetic algorithm          *
*  (1) mateIt, (2) mutateIt, (3) scoreIt, (4) calcAreas, (5) moveEm, (6) getSumAreas     *
* note: getSumAreas uses two other functions sumEm and sumEmIndex                        *
*****************************************************************************************/

/*****************************************************************************************
*                                | function1: mateIt |                                   *
*                                                                                        *
* @purpose creates offspring from a population, generating crossovers according to pCross*
* @param Vs a global array of all the parent and child genomes (Amplitude parameters)    *
* @param ptrs array of pointers from logical indices to actual indices into Vs for       * 
*        each individual                                                                 *
* @param areas the probabilities for choosing each individual for mating                 *
* @param sumArea pointer to the sum of all the individual areas                          *
* @param rands array of random numbers for crossover                                     *         
* @param pCross probability that crossover occurs                                        *               
* @param pSize number of individuals in the population (possible amplitudes solutions)   *
* @param genomeSize number of genes in a genome (number of dihedral * periodicity)       *
*****************************************************************************************/

__global__ void mateIt(float *Vs, int *ptrs, const float *areas, const float *sumArea, 
        const float *rands, const float pCross, const int pSize, const int genomeSize)
{
  /* figure out index for threads  blockId.x is the index for blocks, 
     blockDIM.x is the elements per blocks (# of threads in a block)
     threadIdx is the index for threads */
  int i=blockIdx.x * blockDim.x + threadIdx.x;

  /* random numbers for first parent, second parent, crossover */
  int randi=i*3;

  /* multiply i by 2, as we will have 2 parents and 2 offspring using a left bitwise 
  (<<) by 1*/
  i<<=1;

  /* if we're in the population (sometimes warps may go past) */ 
  if (i<pSize) {
    int parent[2];
    int j;
  /* figure out parents */
    parent[0]=parent[1]=-1;
  /* find parent where cumulative (cum) area (A) is less than random target (tgt) area
    selection of parents depends on cumulative probability being less than the 
    random probabilities (random numbers). rands[] is a function that returns a 
    pseudo-random number (pseudo random number generator)
    The pseudo random numbers range from 0 to RAND_MAX(default is 32767) 
    The random probabilities (tgtA) is random numbers multiply by sum of all the 
    individual probabilities*/
   
    float cumA=0.0f, tgtA=rands[randi++]* *sumArea; //tgtA random number from 0 to the sumArea
    while(cumA<=tgtA){
      ++parent[0];
      cumA+=areas[ptrs[parent[0]]/genomeSize]; // areas is e^-(scorei/score0)
      /* rands[randi-1] is the index back to zero since it is the first set of parents */
    }

#if DEBUG>2
    printf("rands[%d] ; %f ; %f=%f * %f\n",randi, cumA, tgtA, rands[randi-1], *sumArea);
    printf("first parent\n");
#endif

    /* This substract 1st parent area from sum of area  */
    cumA=0.0f; tgtA=rands[randi++]* (*sumArea-areas[ptrs[parent[0]]/genomeSize]); 
    while (cumA<=tgtA){
      ++parent[1];
      if (parent[1]==parent[0])  //Ensure you don't pick the same parents
        ++parent[1];
      cumA+=areas[ptrs[parent[1]]/genomeSize];
    }
#if DEBUG>2
    printf("Make offspring %d from %d and %d (%f=%f*(%f-%f)) %d\n", i, parent[0], 
       parent[1], tgtA, rands[randi-1], *sumArea, areas[ptrs[parent[0]]/genomeSize], randi);
#endif
    /* add offset of pSize to i because it is a child (next population) */
    i+=pSize;
    /* use ptrs to get indices into Vs */
    int i0=ptrs[i], i1=ptrs[i+1];
    parent[0]=ptrs[parent[0]];
    parent[1]=ptrs[parent[1]];
    /* set j to index for the next set of Vs */
    j=i0+genomeSize;
    /* put parent[0], parent[1], and i1 relative to i0, so we can just add i0 for index */
    parent[0]-=i0;
    parent[1]-=i0;
    i1-=i0;
    /* start with crossover pt at the end (no crossover) */
    int crossPt=j;
    /* check if we need to do crossover, 
       only do crossover if random number is less than pCross */
    if(rands[randi]<pCross){
      crossPt=i0+1+(int)(rands[randi]/pCross*(float)(genomeSize-1));
    }
    while(i0<crossPt){
      /* load next bit from parent and increment i */
      Vs[i0]=Vs[parent[0]+i0];
      Vs[i1+i0]=Vs[parent[1]+i0];
      ++i0;
    }
    while(i0<j){
      Vs[i0]=Vs[parent[1]+i0];
      Vs[i1+i0]=Vs[parent[0]+i0];
      ++i0;
    }  //end of while loop
  } // end of if i<pSize loop 
}

/*****************************************************************************************
                                | function 2: mutateIt |

 * @brief introduces mutations to the genomes in Vs, according to probability pMut, 
    with a max perturbation of max
 *
 * @param Vs a global array of all the parent and child genomes
 * @param ptrs array of pointers from logical indices to actual indices into Vs for
     each individual
   @param rands array of random numbers
 * @param pSize number of individuals in the population
 * @param pMut probability that a mutation occurs, evaluated for each gene
 * @param max maximum perturbation to an allele
 * @param genomeSize number of genes in a genome
*******************************************************************************************/

__global__ void mutateIt(float *Vs, int *ptrs, const float *rands, const int pSize, const float pMut, const float max, const int genomeSize, const float *AZP )
{
  /* figure out index */
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<pSize){
    // get index into random number array
    int r=i*genomeSize;
    i=ptrs[i];
    int j=i+genomeSize;
    // want random numbers from [-max, max). will subtract max later
    float scale=2.0f*max/pMut;
    int az = -1;
    int mv;
    int indx;
    // iterate through genome
    while(i<j){
      az++;
      //printf("i is %d, j is %d, az is %d, azp is %f\n", i,j,az,AZP[az]);
      //printf("Vs before random mutation is %f\n", Vs[i]);
      if (AZP[az] < 0.0){
           mv = i + (-AZP[az]);
           Vs[i] =  Vs[mv];
           indx = i;
           //printf("in if: Vs[i] %f, Vs[mv] %f\n", Vs[i],Vs[mv]);
      }
      Vs[mv] = Vs[indx];
      //printf("in if: Vs[mv] %f, Vs[indx] %f\n", Vs[mv],Vs[indx]);
      if(rands[r]<pMut){
         // if Vs equal to zero (if amplitude is zero, leave it as zero)
        if (Vs[i] == 0.0) {
          // Ensure if Vs start off as zero it remain zero]
          //printf("Vs before zero is %f\n", Vs[i]);
          Vs[i] = 0.0;
          //printf("Vs after zero is %f\n", Vs[i]);
        }
        else {
          // mutate the amplitude by adding perturbation based on max, random number and pMut
          //printf("Vs before mutation is %f\n", Vs[i]);
          Vs[i]+=rands[r]*scale-max;
          // if AZP is negative then make Vs for that periodicity equal to the Vs plus the number
          // This is a way to ensure two Vs be the same value
          if (AZP[az] < 0.0){
            mv = i + (-AZP[az]);
            //printf("BEFORE: AZP is %f, mv is %d, i is %d, Vs[i] %f =  Vs[i+mv] %f\n", AZP[az],mv,i,Vs[i],Vs[i+mv]); 
            Vs[i] =  Vs[mv];
            indx = i;
            //printf("AZP is %f, mv is %d, i is %d, Vs[i] %f =  Vs[i+mv] %f\n", AZP[az],mv,i,Vs[i],Vs[i+mv]); 
            //printf("Vs after mutation is %f\n", Vs[i]);
          }
          Vs[mv] = Vs[indx];
          //printf(" and Vs after mutation is %f\n", Vs[i]);
        }
      }
      ++i;
      ++r;
    } // end of while loop
  } 
}

/************************************************************************************************
                                | function 3: scoreIt | 

 * @brief calculates a score indicating the closeness of fit for each individual/chromosome
   (set of parameters) against the training set
 * @param scores score for each conformation, calculated here, output array
 * @param areas weighting for each conformation, no longer need
 * @param Vs a global array of all the parent and child genomes (amplitudes)
 * @param ptrs array of pointers from logical indices to actual indices into Vs for each individual
 * @param tset training set
 * @param tgts targets for training
 * @param wts weights of each point in the training set
 * @param breaks breaks in training set, where different data should not be compared across breaks
 * @param nConf number of conformations in training set
 * @param pSize number of individuals in the population
 * @param genomeSize number of genes in a genome
 * @param xx space to store energy differences for each conformation with test parameters
************************************************************************************************/

__global__ void scoreIt(float *scores, float *areas, const float *Vs, const int *ptrs, const float *tset, const float *tgts, const float *wts, const int *breaks, const int nConf, const int pSize, const int genomeSize, float *xx )
{
  // i represent a chromosome , a set of amplitude parameters, this function will be done for each i (chromosome) at the same time
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<pSize){
    float *x=xx+i*nConf;  // for the error of each conformation
    // get reference to score, S is the AAE 
    float *S=scores+i;
    // set score to 0
    *S=0.0f;
    // accumulate little s for each set
    float s;
    // get first index in genome
    int i0=ptrs[i];
    // get index of next genome space for looping bounds
    int j=i0+genomeSize;
    // start with the first element in the training set
    int t=0;
    /* start at break 0 */
    int b=0;
    /* loop over conformations c */
    int c=0;
    while(c<nConf){
      //s is the sum of REE 
      s=0.0f;
      /* loop only in units without break points */
      while(c<breaks[b+1]){
      /* start with delta E (tgts) for a given conformation (c) within a break; see load.cpp 
           conf (c) goes through until it reach a break. the loop will set delta E */
        x[c]=tgts[c];
        /* subtract contributions from each parameter for conformation c for each conformation 
          e.g deltaE - cos (dihedral * periodicity) * parameter generated from chromosomes 
          Therefore, it is delta E - sum of cosines for each dihedral */
        for(i=i0;i<j;i++,t++){
          x[c]-=Vs[i] * tset[t]; // tset is cos(n * dih)
#if DEBUG>2
          printf("scoreIt: Azp for c%d is %f with az index of %d and target is %f with V of %f\n",c,AZP[az],az,tset[t],Vs[i]);
#endif
          // this one uses V *(1 + cos(n*dih) but it is the same as above
          //x[c]-= (Vs[i] * (1 + tset[t]));
        }
        /* add differences in this error from all other errors */
        for(int c2=breaks[b];c2<c;c2++){
#if DEBUG>2
          printf("%d - %d\n",c,c2); //print the pairs index
#endif
          // calculate the absolute error for each pairs 
          float err=x[c]-x[c2];
          // sum the absolute of the errors (err) - -err = + err ; +err = +err
          //s+=(err<0.0f?-err:err); //ternary operator, condition is err < 0.0; if true err is negative, if false error is positive 
          s+=abs(err); 
        }
        /* next conformation */
        ++c;
      } 
      /* add little error to big error S, weighted by number of pairs, wt  is 2 / nconf*(nconf-1) */
      *S+=s*wts[b];
      /* go to next breakpoint (data set) */
      ++b;
    }
  } //end if in Psize
}

/**************************************************************************************************
*                                 | function 4: calcAreas |                                       *
*                                                                                                 *
*     calculates the areas (the probability) each individual has of mating                        *
*___________________________________Parameters____________________________________________________*
* @param scores scores for each individual (set of parameters)                                    *
* @param areas fitness for each individual, in terms of probability of mating                     *
* @param ptrs array of pointers from logical indices to actual indices into Vs for each individual*
* @param pSize number of individuals in the population                                            *
* @param genomeSize number of genes in a genome                                                   *
**************************************************************************************************/

__global__ void calcAreas(float *scores, float *areas, const int *ptrs, const int pSize, const int genomeSize) {
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  //if((i<<1)<(pSize-1)*pSize){
  if(i<pSize){
    areas[ptrs[i]/genomeSize]=__expf(-scores[i]/scores[0]);
  }
}
/*****************************************************************************************
*                                | function 5: moveEm |
*
* @brief simple helper function for copying data from oldF, oldI to neWF, newI
*
* @param newF pointer to new float array
* @param newI pointer to new int array
* @param oldF pointer to old float array
* @param oldI pointer to old int array
* @param N number of floats/ints to copy
*****************************************************************************************/

__global__ void moveEm(float * newF, int *newI, float *oldF, int *oldI, int N) {
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  if(i<N){
    newF[i]=oldF[i];
    newI[i]=oldI[i];
  }
}
/******************************| function 5 ends |***************************************/

/*****************************************************************************************
                   | sumEm and sumEmIndex : helper function for getSumAreas |

* @brief performs a sum of each successive pair of N numbers in source and stores the sums 
         in sums. intended to be run multiple times to sum over a whole array. if N is odd, 
         the last sum index will be N/2-1 and contain the sum of the last 3 numbers
*
* @param sums where to store the sums
* @param source where to get the numbers to sum together
* @param N the dimension of source
*
* @return                        ********************************************************/

__global__ void sumEm(float *sums, float *source, int N){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=(i<<1);
  if(j+3<N)sums[i]=source[j]+source[j+1];
  else if(j+3==N) sums[i]=source[j]+source[j+1]+source[j+2];
  else if(j+2==N) sums[i]=source[j]+source[j+1];
}

/*
* @brief performs a sum of pairs of N numbers in source, using locations indicated 
         by pointers. pointers has indices multiplied by genomeSize. intended to be
         run multiple times to sum over a whole array. if N is odd, the last sum index 
         will be N/2-1 and contain the sum of the last 3 numbers
*
* @param sums where to store the sums
* @param source an array where to get the numbers to sum together
* @param N the dimension of source
* @param ptrs the indices to use when gathering pairs for summation
* @param genomeSize the number by which the indices in ptrs are scaled
*
* @return 
*/
__global__ void sumEmIndex(float *sums, float *source, int N, const int *ptrs, const int genomeSize){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=(i<<1); // j = i*2 (mutiplication using a left bitwise shift)
  if(j+3<N)sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize];
  else if(j+3==N) sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize]+source[ptrs[j+2]/genomeSize];
  else if(j+2==N) sums[i]=source[ptrs[j]/genomeSize]+source[ptrs[j+1]/genomeSize];
#if DEBUG>1
  if(j+2<=N)printf(" %d:%f",i,sums[i]);
#endif
}
/*******************************| end of helper function |*******************************/
/*****************************************************************************************
*                                | function 6: getSumAreas |                             * 
*                        ---------uses sumEmIndex and sumEM--------                      *
*                                                                                        *
* @brief get sum of all areas                                                            *
* @param areas_d pointer to areas on device                                              *
* @param ptrs_d pointer to indices for each individual in population                     *
* @param pSize population size                                                           *
* @param temp_d pointer to temporary array on device                                     *
* @param genomeSize number of alleles in genome                                          *
*****************************************************************************************/

float *getSumAreas(float *areas_d, int *ptrs_d, int pSize, float *temp_d, const int & genomeSize){
  int dim=pSize; //Set dim to pSize
  int offset=0;

/* The triple chevron below describes an execution configuration the first
    argument(((dim>>1)+BLOCK_SIZE-1)/BLOCK_SIZE) in the execution configuration specifies 
    the  number of thread blocks in the grid, and the second specifies (BLOCK_SIZE) 
     the number of threads in a thread block */

  // return an array of sums (temp_d), sum up the probabilities in areas_d array
  sumEmIndex <<<((dim>>1)+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (temp_d, areas_d, dim, ptrs_d, genomeSize);

#if DEBUG>1
  std::cout << std::endl;
#endif

  pSize >>= 1; // divide pSize by 2 using a right bitwise shift 
  while((dim>>=1)>1){  // while pSize/2 is greater than 1: Keep dividing (1/2 psize) by 2  
    offset^=pSize;  //bitwise XOR offest is 1/2 pSize then 0, then 1/2 pSize, then 0...
    // doing this switch the source to be (temp+pSize/2) then the source changes to (temp_d+0), then back and forth
    sumEm <<<((dim>>1)+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (temp_d+offset, temp_d+(offset^pSize), dim);

#if DEBUG>1
  std::cout << std::endl;
#endif
  }
  return temp_d+offset;
}


/*
///////////////////////////////////////////////////////                 `
//////////////////////////////////                                       `
/////////////////////                                                  |   | 
/////////////                                                     ~ ~ ~ ~ ~ ~ ~
////////                                                         |              |
/////                                                        ____|              |____  
///                                                         |                        | 
//                                                       ___|          J.M           |___
/                                                       |              K.B               |
/                              PROGRAM BEGINS HERE      |              C.T               |
*****************************************************************************************/

/*****************************************************************************************
argc is a vairable with the number of arguments passed to GenA
argv is a vector of strings representing the the arguments the GenA takes
input file: parametersfitting data using the following format:
 _____________________________________________________________________        
|-<dihedral> <AMBER atom type for dihedral 1>                         |
|-<dihedral> <AMBER atom type for dihedral 2>                         |
|<name of data set> <dihedral 1> <dihedral 2>                         |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
|                    ...                                              |
|/                                                                    | 
|<name of data set> <dihedral 1> <dihedral 2>                         |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |
| <dihedral 1 value> <dihedral 2 value> <E_QM> <E_MM>                 |  
|                   ...                                               |
|/                                                                    |  
|_____________________________________________________________________|

<dihedral> is the name of dihedral e.g phi, psi, chi1, chi2, chi3, etc
<AMBER atom type for dihedral 1> e.g chi1 is N -CX-2C-2C for Met, get from frcmod file
<name of data set> is any name, e.g Metalpha, Metbeta, Metcharge
<dihedral 1 value> this is the dihedral value (deg) of the optimized QM structures 
     e.g 105.62
<E_QM> the QM energy of conformation i with restraint dihedral
<E_MM> the MM energy of conformation i with with zeroed dihedral parameters in the 
       frcmod
... repeat for all conformations within a break 
/ (refer to as break (brk))
a break seperate conformations that are different database
    e.g alpha backbone, beta backbone, charge amino acids
                                  GOODLUCK!!!
                                  [ O    O ]
                                  [    b ' ]
                                  [  ----- ]
contact: kellonbelfon@gmail.com with genA title for help
*****************************************************************************************/

int main(int argc, char *argv[]){

  /* start the timer */
  auto t1=std::chrono::high_resolution_clock::now();

  /*specify the string name of the savefile, scorefile, loadfile etc */
  std::string saveFile, loadFile, scoreFile, logFile, frcmodFile, inputFile, fitFile, azpFile;

  /* genetic algorithm parameters initiated */
  int pSize, nGen, rseed, peng, ncp, nCos, nChrom;
  float pMut, max, pCross, keep, nDataset;

  /* getting the filenames from the commands -r, -c, -s, -o, -f -y -a */
  for (int i=1;i<argc;i++){
    if(i+1<argc){
      if(argv[i][0]=='-'&&argv[i][1]=='r')saveFile=argv[++i];  //file that save amplitudes parameter (Vs)
      else if(argv[i][0]=='-'&&argv[i][1]=='c')loadFile=argv[++i]; //file with Vs for restart or from other forcefields
      else if(argv[i][0]=='-'&&argv[i][1]=='s')scoreFile=argv[++i]; // file that save the scores
      else if(argv[i][0]=='-'&&argv[i][1]=='f')frcmodFile=argv[++i]; //file that save frcmod file
      else if(argv[i][0]=='-'&&argv[i][1]=='o')logFile=argv[++i]; //file that save outputs 
      else if(argv[i][0]=='-'&&argv[i][1]=='i')inputFile=argv[++i]; // input file with dihedral info
      else if(argv[i][0]=='-'&&argv[i][1]=='y')fitFile=argv[++i]; // file with and idea of how your target energy change
      else if(argv[i][0]=='-'&&argv[i][1]=='a')azpFile=argv[++i]; // file with the periodicty that amplpitude will be zero
    }
  }

  /* open the output file which is the log file */
  std::ofstream logfile;
  logfile.open (logFile.c_str(), ios::out);
  
  /* open the score file to store scores */
  std::ofstream scorefile;
  scorefile.open (scoreFile.c_str(), ios::out); 
  scorefile << "#Generation" << std::setw(14) << "Chromosomes" << std::setw(12) << "Scores" << std::setw(14) << "areas\n";
 
  /* Now load genA parameters, from the parmfile -p  */
  for (int i=1;i<argc;i++){
    if(i+1<argc){
      if(argv[i][0]=='-'&&argv[i][1]=='p'){
      ConfigFile cfg(argv[++i]); //file that has the genetic algorithm parameters
  
      // check if keys exixt
      if (!(cfg.keyExists("pSize"))) std::cout << "pSize was not specified, using default of 2000\n";  
      if (!(cfg.keyExists("nGen"))) std::cout << "nGen was not specified, using default of 1000\n";  

      // Retreive the value of keys 
      pSize = cfg.getValueOfKey<int>("pSize", 2000);
      logfile << "Population Size (pSize): " << pSize << "\n\n";
      nGen = cfg.getValueOfKey<int>("nGen", 1000);
      logfile << "Number of Generations (nGen): " << nGen << "\n\n";
      pMut = cfg.getValueOfKey<float>("pMut", 1);
      logfile << "Probability of Mutations (pMut): " << pMut << "\n\n";
      max = cfg.getValueOfKey<float>("max", 0.5);
      logfile << "Maximal permissible mutation (max): " << max << "\n\n";
      pCross = cfg.getValueOfKey<float>("pCross", 0.8);
      logfile << "Probability of crossover (pCross): " << pCross << "\n\n";
      rseed = cfg.getValueOfKey<int>("rseed", 314245);
      logfile << "Random seed (rseed): " << rseed << "\n\n";
      peng  = cfg.getValueOfKey<int>("peng", 5);
      logfile << "Print scores every  " << peng << "generations (peng)\n\n";
      ncp  = cfg.getValueOfKey<int>("ncp", 1);
      logfile << "Print scores of only " << ncp << " chromosomes every peng \n\n";
      nCos = cfg.getValueOfKey<int>("nCos", 4);
      logfile << "Periodicity (nCos): " << nCos << "\n\n";
      keep = cfg.getValueOfKey<float>("keep", 0.1);
      logfile << "We will use " << keep << " for the elitist regime\n\n"; 
      nDataset = cfg.getValueOfKey<int>("nDataset", 1);
      logfile << "Number of Dataset is : " << nDataset << "\n\n";
      if(!loadFile.empty()) {
        nChrom = cfg.getValueOfKey<int>("nChrom", 1);
        logfile << "Number of chromosome reported is : " << nChrom << "\n\n";
        }
      }
    } 
  }

 
/* initializing GPU (_d) and CPU arrays */ 
  hipError_t error;
  size_t nRands;
  hiprandGenerator_t gen;
  float *Vs, *Vs_d, *rands, *rands_d, *tset, *tset_d, *tgts, *tgts_d, *wts, *wts_d, *xx_d;
  float *AZP_d, *AZP, *scores, *scores_d, *areas, *areas_d;
  int genomeSize, g, *ptrs_d, *ptrs, N, nConf=0, *breaks, *breaks_d, nBreaks; 
  int save=pSize*keep; //save is number of chromosome we will keep as elitist


/***************************| load data from load.cpp |***********************************
*  check load.cpp for this section                                                       *
*  map is a way to create a dictionary, correction map is an array with key              * 
*****************************************************************************************/

/* initiating container with key and values name correctionMap */
  std::map<std::string,DihCorrection> correctionMap;

/* input file open, with dihedral info */ 
  std::ifstream inputfile;
  inputfile.open (inputFile.c_str(), std::ios::in);
 
/* load in arrays generated from load.cpp, check it out for further comments */
  load(inputfile, &tset, &tgts, &wts, &nConf, &breaks, &nBreaks, &genomeSize, 
       correctionMap, nCos);
  logfile << "Input file loaded ('_')" << "\n\n";
/****************************************************************************************/

/*************************| memory allocation |*******************************************
*   Declare and allocate host and device memory, copy data arrays from CPU host 
       (breaks,tset,                                                                 
*     tgts,wts) to device GPU (breaks_d, etc)                                        
*****************************************************************************************/

#if DEBUG && 0
  for(int i=0;i<nConf;i++){
    for(int j=0;j<genomeSize;j++)
      std::cerr << ' ' << tset[i*genomeSize+j];
    std::cerr << std::endl;
  }
  std::cerr << tgts[0] << ' ' << tgts[1] << ' ' << tgts[2] << ' ' << tgts[3] << std::endl;
  std::cerr << "first hipMalloc, " << nBreaks << " breaks" << std::endl;
#endif

/* Allocate memory on GPU */
  hipMalloc((void **)&breaks_d, nBreaks*sizeof(int));
  hipMalloc((void **)&tgts_d, (nBreaks-1+nConf*(1+genomeSize))*sizeof(float));
  wts_d=tgts_d+nConf;
  tset_d=wts_d+nBreaks-1;

#if DEBUG
  std::cerr << "COPY" << std::endl;
#endif

/* Copying over the arrays from the CPU to GPU
nbreaks is the # of dataset + 1. e.g if you are doing alpha and beta backbone set then nbreaks=3
genomesize is the # of fitting dihedral * periodicity, e.g 3 set of dihedral * 4 periodicity = 12
nconf is the # of conformations you are fitting
tgts is (E_QMi-E_MMi) + (E_MMref-E_QMref) for each conformation, which = nconf, see load.cpp
tset is the cos(dih*periodicity) for 4 periodicity for a dihedral for each conformation
so 20 conf will give tgts of 20 (nconf) * 12 (# of dih * periodicity) = 120 
*/
  hipMemcpy(breaks_d, breaks, nBreaks*sizeof(breaks[0]), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  hipMemcpy(tset_d, tset, nConf*genomeSize*sizeof(float), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  hipMemcpy(tgts_d, tgts, nConf*sizeof(float), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  hipMemcpy(wts_d, wts, (nBreaks-1)*sizeof(*wts), hipMemcpyHostToDevice);
  if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}

/**********************| initiate GPU blocks and # of random variable |*************************** 
*          we need randoms, new pop 3xcrossover, genomeSizexmut                                  *    
*        genome size is the number of genes which is all the parameters,                         *
*   e.g for 4 periodicity and three dihedral fitting, then genomesize will be 4 * 3 = 12         *
*   nRands is number of randoms we need for each set of parameters                               *
*   e.g if psize (population size) is 10, then number of random number we will need is           *
*                   (3+(# of periodicity x # of dihedral)) * psize                               *
* so for 4 periodicity and 3 dihedral fitting (chi1 chi2 chi3), then nRands = 3+12 * 10 = 150    *
*________________________________________________________________________________________________*  
*  nBlocks is dependent on the population size, it is use to figure out how many GPU blocks      *
*  we need to initialize the arrays for calculations. Each block has 256 threads.                *
*  one thread represent one individual (chromosome with soln parameters) from the population     *
*   e.g population size of 2000 will require (2000+256-1)/256 = 8.81 => 8 blocks                 *
*                                                                                                *
*************************************************************************************************/
  nRands=(3+genomeSize)*pSize;
  int nBlocks=(pSize+BLOCK_SIZE-1)/BLOCK_SIZE;

#ifdef DEBUG
  std::cerr << nRands << "nRands\n";
  std::cerr << nBlocks << " blocks\n";
#endif

/*******************************| initializing host and device variables|************************
*         N (bitwise operation below) is the pSize (1st input) multiply by 2;                   *
*       initiating the chromosomes  which have the solns                                        *
************************************************************************************************/
#if DEBUG
  printf("Allocate memory\n");
#endif

  rands=(float *)malloc(nRands*sizeof(float));
  N=(pSize<<1);
  HANDLE_ERROR(hipMalloc((void **)&Vs_d, (N*(genomeSize+4)+pSize*nConf+nRands)*sizeof(float)));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  rands_d=Vs_d+N*genomeSize;
  scores_d=rands_d+nRands;
  areas_d=scores_d+(N<<1);
  xx_d=areas_d+(N<<1);
  scores=(float *)malloc(sizeof(*scores)*N);
  float *scores_ds[2];
  scores_ds[0]=scores_d;
  scores_ds[1]=scores_d+N;

  // allocate memory to host Vs (amplitudes or barrier height for the cosine function)
  Vs=(float *)malloc(N*genomeSize*sizeof(float));
  areas=(float *)malloc(N*sizeof(float));
  /* allocate the memory space to hold array of pointers (prts) of size N (2*pSize)
  these pointers point to the individuals (chromosome) in the population */
  ptrs=(int *)malloc(sizeof(int)*N);
  ptrs[0]=0;
  for(g=1;g<N;g++)ptrs[g]=ptrs[g-1]+genomeSize;
  HANDLE_ERROR(hipMalloc((void **)&ptrs_d, N*2*sizeof(int)));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  int *ptrs_ds[2];
  ptrs_ds[0]=ptrs_d;
  ptrs_ds[1]=ptrs_d+N;
  hipMemcpy(ptrs_d, ptrs, sizeof(int)*N, hipMemcpyHostToDevice);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  int curList=0;

#if 0
  HANDLE_ERROR(hipMalloc((void **)&scores_d, N*sizeof(float)));

  HANDLE_ERROR(hipMalloc((void **)&xx_d, nOffspring*nConf*sizeof(float)));
#endif

/* thrust is a c++ template library for CUDA similar to STL it have two containers: 
       thrust::host_vector<type> and thrust::device_vector<type>
  The containers make common operations such as hipMalloc, hipFree, hipMemcpy, more concise
  e.g thrust::host_vector<int> vec_h(2) will allocate host vector with 2 elements
    thrust::device_vectore<int> vec_d = vec_h will copy host vector to device
  This will allow you to directly manipulate device values from the host
    so vec_d[0] = 5; can be done from host  and once you output vector memory is 
    automatically released 
   it have a few algorithms, we use thrust::sort(), */
  thrust::device_ptr<int> dPtrs(ptrs_d), dPtrs_save(ptrs_d+save);
  thrust::device_ptr<float> dScores(scores_d), dVs(Vs_d);
  thrust::device_ptr<float> dScores_save(scores_d+save),
                            dScores_pSize(scores_d+pSize),
                            dScores_N(scores_d+N);


/**************************| Create a random generator |********************************************
* hiprandCreateGenerator takes two parameters: pointer to generator (*gen), type of generator       *
Once created,random number generators can be defined using the general options seed, offset,& order*
When rng_type is HIPRAND_RNG_PSEUDO_DEFAULT, the type chosen is HIPRAND_RNG_PSEUDO_XORWOW            *
*__________________________________________________________________________________________________*
*hiprandSetPseudoRandomGeneratorSeed takes two parameters (1) the generator (gen) & (2) seed value  *
* seed value # is used to initialize the generator and control the set of random numbers;          *
* same seed will the give same set of random numbers of the psuedorandom generator                 *
* rseed is the random number specified from the 6th input)                                         *
*__________________________________________________________________________________________________*
*    hiprandGenerateNormal take 5 parameters:                                                       * 
*  (1) generator - Generator to use                                                                *
*  (2) outputPtr - Pointer to device memory to store CUDA-generated results,                       *
                or Pointer to host memory to store CPU-generated resluts                           *
*  (3) num - Number of floats to generate                                                          *
*  (4) mean - Mean of normal distribution                                                          *
*  (5) stddev - Standard deviation of normal distribution                                          *
* Results are 32-bit floating point values with mean and standard deviation.                       * 
***************************************************************************************************/

#if DEBUG
  printf("Create random generator\n");
#endif

  // create the generator name gen
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

#if DEBUG
  printf("Seed random generator\n");
#endif
  // initiate the generator with the random seed (rseed)
  hiprandSetPseudoRandomGeneratorSeed(gen, rseed);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (seed)\n", hipGetErrorString(error));}
#if DEBUG
   std::cerr << "GenerateNormal" << std::endl;
#endif
  // Vs_d is the amplitudes which is random numbers
  hiprandGenerateNormal(gen, Vs_d, N*genomeSize, 0, 1);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (normal)\n", hipGetErrorString(error));}

#if DEBUG
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
  /// print the three Vs from the first two chromosomes. 
  std::cout << "random Vs, created on GPU" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
#endif

  /*****  if we have a load file copy Vs (amplitude parameters) from the loaded file and populate Vs ***********/
  if(!loadFile.empty()) {
    std::ifstream loadfile;
    loadfile.open (loadFile.c_str(), std::ios::in);
    // copy the random Vs to add previous chromosome of nChrom
    hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
    if (loadfile.is_open()) {
      for (int i=0;i<nChrom;i++) {
        for (int j=0;j<genomeSize;j++) {
          loadfile >> Vs[ptrs[i]+j]; 
        }
      }
    }
    // print the three Vs from the first two chromosomes, to ensure your Vs were loaded. 
    logfile << "Here is your loaded Vs(amplitudes) for first two chromosomes: \n\n" << std::endl;
    for(int i=0;i<2;i++){
      for(int j=0;j<genomeSize;j++){
        logfile <<  Vs[ptrs[i]+j] << " ";  
      }
      logfile << "\n";
    }      

    // copy loaded Vs  to the GPU and overwrite random Vs. If user only create two chromosomes or 
    // previous Vs then the rest of the chromosome will be random 
    hipMemcpy(Vs_d, Vs, N*genomeSize*sizeof(*Vs), hipMemcpyHostToDevice);// copy to GPU 
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
  }

#if DEBUG
  // check to see if Vs was transfer to gpu successful 
  /// print the three Vs from the first two chromosomes. 
  std::cout << "loaded Vs" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
  hipMemcpy(Vs_d, Vs, pSize*genomeSize*sizeof(*Vs), hipMemcpyHostToDevice);// copy to GPU 
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost); // copy back to CPU
  /// print the three Vs from the first two chromosomes. 
  std::cout << "After transfer of loaded Vs to GPU" << std::endl;
  for(int i=0;i<1;i++){
    std::cout <<  Vs[ptrs[i]] << " " << Vs[ptrs[i]+1] << " " << Vs[ptrs[i]+2] << std::endl;  
  }    
#endif
  
   /*  This section zero amplitudes that we want based on the AZP file  */
  // if we have AZP file we will read in the values and store it in AZP arrays
  // AZPfile has to have a number of elements equal to genomsize 
  if(!azpFile.empty()) {
    // allocate memory for AZP to zero amplitudes, This array is multiply by Vs   
    AZP=(float *)malloc(genomeSize*sizeof(float));
    std::ifstream azpfile;
    azpfile.open (azpFile.c_str(), std::ios::in);
    if (azpfile.is_open()) {
      for(int j=0;j<genomeSize;j++){
        azpfile >> AZP[j]; 
      }
    }
    HANDLE_ERROR(hipMalloc((void **)&AZP_d, (genomeSize)*sizeof(float)));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
 
    // now apply AZP, first copy over Vs from GPU
    hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
    for(int i=0;i<N;i++){
      int az = 0;
      for(int j=0;j<genomeSize;j++,az++){
        // Because some of the Vs is negative let us ensure Vs become 0. If az is 1 then nothing Vs remain
#ifdef DEBUG
        printf("Before Vs is (%f) and azp is %f\n", Vs[ptrs[i]+j], AZP[az]);
#endif
        // if AZP is 0 then the amplitude will be zero
        if (AZP[az] == 0.0){
          Vs[ptrs[i]+j] = 0.00000; 
        }
        // if AZP is negative then make Vs for that periodicity equal to the Vs plus the number
        // This is a way to ensure two Vs be the same value 
        if (AZP[az] < 0.0){
          int mv = j + (-AZP[az]);
          Vs[ptrs[i]+j] =  Vs[ptrs[i]+mv];
        }
#ifdef DEBUG
        printf("az index is %d, AZP is %f, mv is %d, i is %d, Vs[i] %f =  Vs[i+mv] %f\n", az,AZP[az],mv,i,Vs[ptrs[i]+j],Vs[ptrs[i]+mv]); 
        printf("After Vs is (%f)\n", Vs[ptrs[i]+j]);
        printf("Vs is %f for az%d of AZP value %f for genome %d and psize %d\n",  Vs[ptrs[i]+j],az,AZP[az],j,i);
#endif
      }
    }      
    // copy new Vs to GPU, need to clean up the code (too much copying, only copy Vs once) 
    hipMemcpy(Vs_d, Vs, sizeof(float)*genomeSize*N, hipMemcpyHostToDevice);// copy to GPU 
    hipMemcpy(AZP_d, AZP, sizeof(float)*genomeSize, hipMemcpyHostToDevice);// copy to GPU 
  } //end AZP section  

#ifdef DEBUG
  for(int j=0;j<genomeSize;j++){
    printf("aZp is %f\n",AZP[j]);
  }
#endif

 
/***************************| score of the first set of chromosomes |*******************************
* Here we score the two arrays of parents with solution parameters in the initial population       * 
*****************************************nm  *******************************************************/
#if DEBUG
    std::cerr << "1stscore" << std::endl;
#endif

    /* lauch first kernel to score the initial set of chromsomes (Vs_d) and output scores in scores_ds
      betweem the triple chervon is called the execution configuration that takes two parts
      1st part takes the number of thread blocks and the second part take the number of threads in a block */
    scoreIt <<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList], areas_d, Vs_d, ptrs_ds[curList], tset_d, tgts_d, wts_d, breaks_d, nConf, pSize, genomeSize, xx_d);
    /* score of chromosomes out of psize since we initiated 2 times psize */
    scoreIt <<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList]+pSize, areas_d, Vs_d, ptrs_ds[curList]+pSize, tset_d, tgts_d, wts_d, breaks_d, nConf, pSize, genomeSize, xx_d);
  
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (1stscore)\n", hipGetErrorString(error));}

#if DEBUG
    std::cerr << "1stsort" << std::endl;
#endif

    /* sort the scores from each chromosome of the initial population */
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]), thrust::device_pointer_cast(scores_ds[curList]+N), thrust::device_pointer_cast(ptrs_ds[curList]));
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (1stsort)\n", hipGetErrorString(error));}

    // print the initial scores based on ncp as -1 
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*ncp, hipMemcpyDeviceToHost);
    for(int m=0;m<ncp;m++){
      scorefile << std::setw(6) << "-1" << std::setw(14) << m << std::setw(18) << scores[m]/nDataset << "\n";
      }


#if DEBUG>2
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    hipMemcpy(Vs, Vs_d, sizeof(*Vs)*N*genomeSize, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    hipMemcpy(ptrs, ptrs_ds[curList], sizeof(*ptrs)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
       /* i is each chromosome, scores[i] is scores, Vs[ptrs[i]] is the amplitude parameters;
         Vs[ptrs[i]]+n specifies the next n amplitude. e.g chromosome i have genomesize amplitude parms 
         e.g  Vs[ptrs[i]]+1 is the amplitude term when the periodicity is 3 for the 1st dihedral being
        fitted, and  Vs[ptrs[i]]+4, the amplitude term when the periodicity is 4 for the 2nd dihedral */
    for(int i=0;i<N;i++){
      std::cerr << i << ": [" << ptrs[i] << "] = " << scores[i] << " {"<<Vs[ptrs[i]]<<" "<<Vs[ptrs[i]+1]<<" "<<Vs[ptrs[i]+2]<<" "<<Vs[ptrs[i]+3]<<"}\n";
    }
#endif

/****************************| Let us begin the iterations through generations |********************

 Genetic algorithm iterations through the number of generations (nGen: 2nd input) 

****************************************************************************************************/

  /* for loop for the generation */
  for(g=0;g<nGen;g++){
  
/*************************| Step1: Generate random numbers |****************************************/
#if DEBUG>1
  printf("Generate random numbers\n");
  printf(" %d",g);fflush(stdout);
#endif
    
    // create an array of random numbers (rands_d) used for mutations and crossover where the number of random #s is nRands 
    hiprandGenerateUniform(gen, rands_d, nRands);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}

/***| Step2: calculate the probabilities (areas) each individual (chromosome) has of mating |******/
#if DEBUG>2
    std::cerr << "Mate" << std::endl;
#endif
    calcAreas <<<nBlocks, BLOCK_SIZE>>> (scores_ds[curList], areas_d, ptrs_d, pSize, genomeSize);

/***| Step3:  mate the individuals (chromosomes,Parent[0],[1]) selected for the next generation |***/
    mateIt <<<nBlocks, BLOCK_SIZE>>> (Vs_d, ptrs_ds[curList], areas_d, 
  getSumAreas(areas_d, ptrs_ds[curList], pSize, areas_d+N, genomeSize),
 rands_d, pCross, pSize, genomeSize);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (mate)\n", hipGetErrorString(error));}

/*****************| Step4: mutate individuals generated after mating |*****************************/
#if DEBUG>2
    std::cerr << "Mutate" << std::endl;
#endif
    mutateIt <<<nBlocks, BLOCK_SIZE>>> (Vs_d, ptrs_ds[curList]+pSize, rands_d+pSize*3, pSize, pMut, max, genomeSize, AZP_d);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (mutate)\n", hipGetErrorString(error));}

/**************| Step5: Score the individuals to select for the next generation |*******************/
#if DEBUG>2
    std::cerr << "Score" << std::endl;
#endif
    scoreIt <<<nBlocks, BLOCK_SIZE>>> (scores_ds[curList]+pSize, areas_d, Vs_d, ptrs_ds[curList]+pSize, tset_d, tgts_d, wts_d, breaks_d, nConf, pSize, genomeSize, xx_d);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s (score)\n", hipGetErrorString(error));}

#if DEBUG>2
    //std::cerr << "Display em:\n\tCopy scores" << std::endl;
    hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    //std::cerr << "\tCopy Vs" << std::endl;
    hipMemcpy(Vs, Vs_d, sizeof(*Vs)*N*genomeSize, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    //std::cerr << "\tCopy ptrs" << std::endl;
    hipMemcpy(ptrs, ptrs_ds[curList], sizeof(*ptrs)*N, hipMemcpyDeviceToHost);
    if((error=hipGetLastError())!=hipSuccess){fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(error));}
    for(int i=0;i<N;i++){
      /* below you can print the scores for a chromosomes every generation */
      std::cout << "This is Generation: " << g << " and Chromosome (set of parameters): " << i << std::endl;
      std::cout << "Score: " << scores[i] << std::endl;
    /* below you can print out the scores and the first four barrier parameters,since we are using 
     4 periodicity, the first 4 barrier parameters are for the 1st dihedral in the input file */
     std::cerr << i << ": [" << ptrs[i] << "] = " << scores[i] << " {"<<Vs[ptrs[i]]<<" "<<Vs[ptrs[i]+1]<<" "<<Vs[ptrs[i]+2]<<" "<<Vs[ptrs[i]+3]<<"}\n";
    }
#endif

/*****| Step6: Sort the scored chromosomes (individuals) & select for mating for next generation |**/
#if DEBUG>2
    std::cerr << "Move 1" << std::endl;
#endif
    moveEm <<<(save+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1], ptrs_ds[curList^1], scores_ds[curList], ptrs_ds[curList], save);
#if DEBUG>2
    std::cerr << "Move 2" << std::endl;
#endif
    moveEm <<<(pSize+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1]+save, ptrs_ds[curList^1]+save, scores_ds[curList]+pSize, ptrs_ds[curList]+pSize, pSize);//nOffspring);
#if DEBUG>2
    std::cerr << "Move 3" << std::endl;
#endif
    moveEm <<<(pSize-save+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>> (scores_ds[curList^1]+save+pSize, ptrs_ds[curList^1]+save+pSize, scores_ds[curList]+save, ptrs_ds[curList]+save, pSize-save);
    curList^=1;

    /* first sort only the non elitist with the offspring */
#if DEBUG>1
    std::cerr << "Selection sort (" << N << " items, less " << save << ")" << std::endl;
#endif
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]+save), thrust::device_pointer_cast(scores_ds[curList]+pSize+save), thrust::device_pointer_cast(ptrs_ds[curList]+save));

    /* second sort  is to order the first sort within psize by fitness */
#if DEBUG>1
    std::cerr << "Rank sort" << std::endl;
#endif
    thrust::sort_by_key(thrust::device_pointer_cast(scores_ds[curList]), thrust::device_pointer_cast(scores_ds[curList]+pSize), thrust::device_pointer_cast(ptrs_ds[curList]));

/****************************************************************************************************
* Here you can print the score of chromosomes (total is 2 x population size) to score file (-s)     *
****************************************************************************************************/
    //peng --> print every n generation
    //ncp --> number of chromosomes to print
    //if generation is divisable by peng
    if(g%peng==0) {
      //scorefile << "#Generation" << std::setw(14) << "Chromosomes" << std::setw(12) << "Scores\n";
      hipMemcpy(scores, scores_ds[curList], sizeof(*scores)*ncp, hipMemcpyDeviceToHost); //copy over ncp scores
      hipMemcpy(areas, areas_d, sizeof(*areas)*ncp, hipMemcpyDeviceToHost); //copy over ncp areas
      // divide score by the number of datasets to print the average of the datasets since score is sum of each dataset score
      for(int m=0;m<ncp;m++){
        scorefile << std::setw(6) << g << std::setw(14) << m << std::setw(18) << scores[m]/nDataset << std::setw(18) << areas[m] << "\n";
      }
    }

/* END GENETIC ALGORITM */
  } 

  scorefile.close();
/****************************************************************************************************
*    TERMINATION, LAST RESULTS < SCORES AND PARAMETERS FOR EACH INDIVIDUAL
****************************************************************************************************/
 
/***************************************************************************************************/
  /*  copy over the results from GPU to the CPU to save the scores and parameters */
  hipMemcpy(Vs, Vs_d, sizeof(float)*genomeSize*N, hipMemcpyDeviceToHost);
  hipMemcpy(ptrs, ptrs_ds[curList], sizeof(int)*N, hipMemcpyDeviceToHost);
  hipMemcpy(scores, scores_ds[curList], sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(tgts, tgts_d, sizeof(float)*nConf, hipMemcpyDeviceToHost);
  hipMemcpy(tset, tset_d, nConf*genomeSize*sizeof(float), hipMemcpyDeviceToHost);

/****************************************************************************************************/

// Here I am writing out the initial dE and the final dE, see load.cpp for description 
  /* file that stores initial dE */
  std::ofstream fitfile;
  fitfile.open (fitFile.c_str(), ios::out); 
  fitfile << "#dE = (E (QMi) - E (QMref) ) - ( E (MMi) - E (MMref) \n\n";
  fitfile << "#ref is the first structure in each dataset  \n\n";
  fitfile << "#Below is the target relative energy and the best relative energy after the fit \n\n";
  fitfile << "#Conformations" << "      "<< "dE" <<  "      " << "dE_from_fit " <<   "      " << "dE - dE_fromfit" << "\n";
  
  /*  save the last dE  */
  float new_dih[nConf]; // hold the new dihedral(s) contribution 
  for(int i=0;i<nConf;i++){ //loop through the conformations 
    float sum_cos = 0.0f; //array that hold the sum the cosine for each conformation 
    for(int j=0;j<genomeSize;j++){
       // Vs[ptrs[0]+j] this is indexing through best chromosome amplitudes (Vs[ptrs[0]])
       // multiply the amplitude (Vs) by (1+ cos(n*dih) which is (tset[i*genomeSize+j] )
      //printf("Vs : %f\n", Vs[ptrs[i]+j]);
      sum_cos += (Vs[ptrs[0]+j] * (1 + tset[i*genomeSize+j]));
      //printf("cos(n*dih) : %f\n", tset[i*genomeSize+j]);
    }
    new_dih[i] = sum_cos; //Effnew new dihedral energy for each conf
  }

  int b = 0; // b is number of dataset
  int c = 0; // c is conformations
  while(c<nConf){ // check through the conformations 
    float conf0 =  new_dih[breaks[b]]; // conf0 is the 1st conf in the dataset (separated by breaks)
    while(c<breaks[b+1]){ // loop through the dataset
      // printf("1st conf: %f\n", conf0);
      new_dih[c] -= conf0; //subtract to 1st conf to get relative energy
      // printf("newdih: %f\n", new_dih[c]);
      ++c; // go to next conformation
      }
  ++b; // go to next data set
  }  
  for(int i=0;i<nConf;i++){ // loop to save to the fit file
    float temp = tgts[i] - new_dih[i];
    fitfile << std::setw(6) << i << std::setw(18) << tgts[i] << std::setw(16) << new_dih[i] << std::setw(18) << temp << "\n";
  }
  fitfile.close();
/****************************************************************************************************/

  /* saving all of the scores, with dihedral parameters to the logfile */
  logfile << "\n";
  logfile << "Printing all of the final dihedral parameters, check your -f file for the best one \n\n";
  logfile << "The first one is the best score, best parameters\n\n";
  /* loop through the population */
  for(int i=0;i<pSize;i++){
    /* these are the final scores for each individual in the population, print in the output file  */
    // divide score by the number of datasets to print the average of the datasets since score is sum of each dataset score
    logfile << std::fixed << scores[i]/nDataset << std::endl;
    for(std::map<std::string,DihCorrection>::iterator it=correctionMap.begin(); it!=correctionMap.end(); ++it){
    /* second.setGenome(Vs+ptrs[i]) is the dihedral parameters for each individual in the population 
      print in the output file                                                                     */
      logfile << it->second.setGenome(Vs+ptrs[i]);
    }
  }

/****************************************************************************************************/
  /* Save a frcmod file to use in Amber */
  if(!frcmodFile.empty()){
    std::ofstream frcmodfile;
    frcmodfile.open (frcmodFile.c_str(), ios::out);
    frcmodfile << "frcmod from GenA.cu \n";
    frcmodfile << "DIHE\n";
    for(std::map<std::string,DihCorrection>::iterator it=correctionMap.begin(); it!=correctionMap.end(); ++it){
      frcmodfile << it->second.setGenome(Vs+ptrs[0]); //the best parameters 
      }
    frcmodfile.close();
  }

/****************************************************************************************************/
  /* Save the amplitudes to a restart file  */
  if(!saveFile.empty()){
    std::ofstream savefile;
    savefile.open (saveFile.c_str(), ios::out);
    for(int i=0;i<N;i++){
      for(int j=0;j<genomeSize;j++){
        savefile << std::setw(9) << Vs[ptrs[i]+j] << " ";
      }
      savefile <<"\n";
    }
  savefile.close();
  }

/****************************************************************************************************/
#if 0
  std::cout << scores[pSize] << std::endl;
  for(std::map<std::string,DihCorrection>::iterator it=correctionMap.begin(); it!=correctionMap.end(); ++it){
    std::cout << it->second.setGenome(Vs+ptrs[pSize]);
    //std::cout << it->second;
  }
#endif


#if 0
  printf("Copy random numbers\n");
  hipMemcpy(rands, rands_d, nRands*sizeof(unsigned int), hipMemcpyDeviceToHost);
  printf("Print random numbers\n");
  printf("%d",rands[0]);
  for(i=1;i<nRands;i++){
    printf(" %d",rands[i]);
  }
  putchar('\n');
#endif

  auto t2=std::chrono::high_resolution_clock::now();

  logfile <<"\n\n";
  logfile << "GenA took " 
          << std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count() 
          << " milli seconds to obtain your parameters" << "\n";

  logfile.close(); //close log file

/*****************| Free up Memory |*******************************************************/
  free(ptrs);
  hiprandDestroyGenerator(gen);
  //hipFree(xx_d);
  hipFree(Vs_d);
  hipFree(ptrs_d);
  hipFree(breaks_d);
  hipFree(tgts_d);
  free(Vs);
  free(scores);
  //hipFree(rands_d);
  free(rands);
  return 0;
}
